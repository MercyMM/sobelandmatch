#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

#include "cv.h"
#include "highgui.h"
#include "elas.h"
#include <vector>
#include "triangle.h"
#include "matrix.h"
#include <stdlib.h>

using namespace std;

/***
 * 1. hipHostAlloc and hipHostFree wrap
 * 2. create Descriptor of two img
 * 3. compute support point
 * 4. convert dis to cloud
 * 5. cuda_computeD
 * */



#define WIDTH 320
#define HEIGH 240
#define GRID_SIZE 20
enum setting { ROBOTICS, MIDDLEBURY };

// parameter set
Elas::parameters param;


struct plane {
    float   t1a, t1b, t1c;
    float   t2a;
};

/**
 * 1. hipHostAlloc and hipHostFree wrap
 *
 * */
static int count_alloc = 0;
static int count_free = 0;
void cudaFreeHost_cpuaa(void *p)
{
    count_free++;
    hipHostFree(p);
    hipError_t err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));
}

void SetDeviceMap()
{
    hipSetDeviceFlags(hipDeviceMapHost);
}

void* HostMal(void **p, long size)
{
    count_alloc++;
    void *p_g;
//    hipHostAlloc((void**)p,size, hipHostMallocDefault | hipHostMallocMapped);
    hipHostAlloc((void**)p, size, hipHostMallocDefault );
    hipError_t err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));
    //将常规的主机指针转换成指向设备内存空间的指针
    hipHostGetDevicePointer(&p_g, *p, 0);
    err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));
    return p_g;
}

void allocFreeCount()
{
    printf("count_alloc = %d\n", count_alloc);
    printf("count_free= %d\n", count_free);
}

/**
 * 1. over: hipHostAlloc and hipHostFree wrap
 * */

//dim3 threads(320 - 6 , 1);
//dim3 grid( 1, 240 -6 ); => (0, 233)



/**
 * 2. create Descriptor of two img
 * */
__global__ \
void createDesc_gpu_kernel(uint8_t* I_desc, uint8_t* I_du, uint8_t* I_dv)
{

    int u = blockDim.x * blockIdx.x + threadIdx.x;
    int v = blockDim.y * blockIdx.y + threadIdx.y;

    int x = u + 3;
    int y = v + 3;

    __shared__ uint8_t      I_du_share[320 * 5];
    __shared__ uint8_t      I_dv_share[320 * 3];

    uint8_t *I_desc_curr;

    for(int i = 0; i < 5; i++){
        *(I_du_share + x + i * 320) = *(I_du + x + (y-2 + i) * 320);
    }

    for(int i = 0; i < 3; i++){
        *(I_dv_share + x + i * 320) = *(I_dv + x + (y-1 + i) * 320);
    }

    __syncthreads();

    I_desc_curr = I_desc + (y* WIDTH + x) * 16;

    *(I_desc_curr++) = *(I_du_share + (0 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_du_share + (1 * WIDTH + x - 2) );
    *(I_desc_curr++) = *(I_du_share + (1 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_du_share + (1 * WIDTH + x + 2) );

    *(I_desc_curr++) = *(I_du_share + (2 * WIDTH + x - 1) );
    *(I_desc_curr++) = *(I_du_share + (2 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_du_share + (2 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_du_share + (2 * WIDTH + x + 2) );

    *(I_desc_curr++) = *(I_du_share + (3 * WIDTH + x - 2) );
    *(I_desc_curr++) = *(I_du_share + (3 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_du_share + (3 * WIDTH + x + 2) );
    *(I_desc_curr++) = *(I_du_share + (4 * WIDTH + x + 0) );

    *(I_desc_curr++) = *(I_dv_share + (0 * WIDTH + x + 0) );
    *(I_desc_curr++) = *(I_dv_share + (1 * WIDTH + x + 1) );
    *(I_desc_curr++) = *(I_dv_share + (1 * WIDTH + x - 1) );
    *(I_desc_curr++) = *(I_dv_share + (2 * WIDTH + x + 0) );

}

int __createDesc_gpu(uint8_t* I_desc, uint8_t* I_du_g, uint8_t* I_dv_g )
{
    dim3 threads(WIDTH - 6 , 1);
    dim3 grid( 1, HEIGH - 6 );
//    hipDeviceSynchronize();
    createDesc_gpu_kernel<<<grid, threads, 0 >>>(I_desc, I_du_g, I_dv_g );
//    hipError_t err = hipGetLastError();
//    printf("cuda error: %s\n", hipGetErrorString(err));
//    hipDeviceSynchronize(); //2.88 - 0.19

}

/**
 * 2. over: create Descriptor of two img
 * */



/**
 * 3. compute support point
 * */
__device__ \
uint32_t getAddressOffsetImage1(const int32_t& u, const int32_t& v, const int32_t& width)
{
    return v*width + u;
}

__device__ \
unsigned int computeMatchEnergy1(unsigned char* dst1, unsigned char* dst2, int offset)
{
    unsigned int a, b, c, e, r0, r4;

    a = abs(*(dst1 + offset) - *(dst2 + offset)) + abs(*(dst1 + offset + 1) - *(dst2 + offset + 1));
    b = abs(*(dst1 + offset + 2) - *(dst2 + offset + 2)) + abs(*(dst1 + offset + 3) - *(dst2 + offset + 3));
    c = abs(*(dst1 + offset + 4) - *(dst2 + offset + 4)) + abs(*(dst1 + offset + 5) - *(dst2 + offset + 5));
    e = abs(*(dst1 + offset + 6) - *(dst2 + offset + 6)) + abs(*(dst1 + offset + 7) - *(dst2 + offset + 7));
    r0 = a + b + c + e;

    a = abs(*(dst1 + offset + 8) - *(dst2 + offset + 8)) + abs(*(dst1 + offset + 9) - *(dst2 + offset + 9));
    b = abs(*(dst1 + offset + 10) - *(dst2 + offset + 10)) + abs(*(dst1 + offset + 11) - *(dst2 + offset + 11));
    c = abs(*(dst1 + offset + 12) - *(dst2 + offset + 12)) + abs(*(dst1 + offset + 13) - *(dst2 + offset + 13));
    e = abs(*(dst1 + offset + 14) - *(dst2 + offset + 14)) + abs(*(dst1 + offset + 15) - *(dst2 + offset + 15));
    r4 = a + b + c + e;

    return r0 + r4;
}


__device__ \
uint32_t getAddressOffsetGrid1(const int32_t& x, const int32_t& y, \
                                          const int32_t& d, const int32_t& width, const int32_t& disp_num)
{
    return (y*width + x)*disp_num + d;
}


__device__ \
void updatePosteriorMinimumNew(unsigned char* dst1, unsigned char* dst2, const int32_t &d, int32_t &val, int32_t &min_val, int32_t &min_d)
{
    val = computeMatchEnergy1(dst1, dst2, 0);
    if (val<min_val) {
        min_val = val;
        min_d = d;
    }
}

__device__ \
void updatePosteriorMinimumNew1(unsigned char* dst1, unsigned char* dst2, const int32_t &d, const int8_t w, int32_t &val, int32_t &min_val, int32_t &min_d)
{
    val = computeMatchEnergy1(dst1, dst2, 0) + w;
    if (val<min_val) {
        min_val = val;
        min_d = d;
    }
}

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ \
unsigned int computeMatchEnergy1_new(unsigned char* dst1_1, unsigned char* dst1_2, unsigned char* dst2_1, unsigned char* dst2_2, int32_t u, int32_t u_wrap) {
    unsigned int r0, r1, r2, r3;
    r0 = 0;
    r1 = 0;
    r2 = 0;
    r3 = 0;

#pragma unroll
    for (int i = 0; i < 16; i++) {
        r0 += abs(dst2_1[(u_wrap << 4) - 32 + i] - dst1_1[(u << 4) - 32 + i]);
        r1 += abs(dst2_1[(u_wrap << 4) + 32 + i] - dst1_1[(u << 4) + 32 + i]);
        r2 += abs(dst2_2[(u_wrap << 4) - 32 + i] - dst1_2[(u << 4) - 32 + i]);
        r3 += abs(dst2_2[(u_wrap << 4) + 32 + i] - dst1_2[(u << 4) + 32 + i]);
    }

    return r0 + r1 + r2 + r3;
}


#define D_candidate_stepsize 5
#define INCON_THRESHOLD 5
#define INCON_MIN_SUPPORT 5
#define INCON_WINDOW_SIZE 5
#define SUPPORT_TEXTURE 10
#define DISP_MIN 0
#define DISP_MAX 63

#define SUPPORT_THRESHOLD 0.85


#define U_STEP 2
#define V_STEP 2
#define WINDOW_SIZE 3
#define MIN_1_E 32767
#define MIN_1_D  -1
#define MIN_2_E 32767
#define MIN_2_D  -1
#define DESC_OFFSET_1  (-16 * U_STEP)
#define DESC_OFFSET_2  (+16 * U_STEP)
#define DESC_OFFSET_3  (-16 * U_STEP)
#define DESC_OFFSET_4  (+16 * U_STEP)


#define BLOCKX  60
#define BLOCKY  1
#define GRIDX   1
#define GRIDY   46
//#define GRIDY   2
//dim3 threads(60, 1);
//dim3 grid(1, 46);

__constant__ uint32_t oneLine = WIDTH * 16;

__global__ void compEner_gpu(uint8_t* I1_desc_shared, uint8_t* I2_desc_shared, int u, int u_wrap,  uint32_t* sumResult)
{

    int x = threadIdx.x; // x = (0,15)
    int32_t sum = 0;
    sum  = abs(I1_desc_shared[(u - 2) << 4 + x ] - I2_desc_shared[(u_wrap - 2) << 4 + x]);
    sum += abs(I1_desc_shared[(u + 2) << 4 + x ] - I2_desc_shared[(u_wrap + 2) << 4 + x]);
    sum += abs(I1_desc_shared[(u + 2) << 4 + x  + oneLine] - I2_desc_shared[(u_wrap + 2) << 4 + x +oneLine]);
    sum += abs(I1_desc_shared[(u - 2) << 4 + x  + oneLine] - I2_desc_shared[(u_wrap - 2) << 4 + x +oneLine]);
    sumResult[x] = sum;
}

__global__ void sptMathKernel(int32_t D_can_width, int32_t D_can_height, int8_t* D_can, uint8_t* desc1, uint8_t* desc2)
{
    int32_t u_wrap;
    int disp_max_valid;
    int result1 = 0, result2 = 0, result3 = 0, result4 = 0;
    int32_t  line_offset;
    uint8_t *I1_line_addr, *I2_line_addr, *I1_block_addr, *I2_block_addr, *I_line_addr_tmp;
    uint8_t *I1_block_addr_1, *I1_block_addr_2, *I2_block_addr_1, *I2_block_addr_2;
    int32_t sum = 0;
    int16_t min_1_E;
    int16_t min_1_d;
    int16_t min_2_E;
    int16_t min_2_d;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int u, v, d1 = -1 , d2 = -1;

    u = (x + 3) * D_candidate_stepsize;  //5
    v = (y + 1) * D_candidate_stepsize;
    line_offset = 16 * WIDTH*v;
    I1_line_addr = desc1 + line_offset;
    I2_line_addr = desc2 + line_offset;

    __shared__ uint8_t I1_desc_shared[320 * 16 * 2];
    __shared__ uint8_t I2_desc_shared[320 * 16 * 2];

    for(int i = 0; i < 85; i++){
        I1_desc_shared[x + i * BLOCKX ] = *(I1_line_addr + x + i * BLOCKX - 2 * oneLine);
        I1_desc_shared[x + i * BLOCKX + oneLine] = *(I1_line_addr + x + i * BLOCKX + 2 * oneLine);

        I2_desc_shared[x + i * BLOCKX ] = *(I2_line_addr + x + i * BLOCKX - 2 * oneLine);
        I2_desc_shared[x + i * BLOCKX + oneLine] = *(I2_line_addr + x + i * BLOCKX + 2 * oneLine);
    }

    __syncthreads();

    for (int32_t i=0; i<16; i++)
        sum += abs((int32_t)(*(I1_line_addr + 16 * u +i))-128);
    if (sum<10){
      d1 = -1;
      return;
    }

    I1_block_addr_1 = I1_desc_shared + 16 * u ;
    I1_block_addr_2 = I1_desc_shared + 16 * u + oneLine ;

    disp_max_valid = min(63, u - 5);
    min_1_E = MIN_1_E;
    min_1_d = MIN_1_D;
    min_2_E = MIN_2_E;
    min_2_d = MIN_2_D;


    for (int16_t d = 0; d <= disp_max_valid; d++) {
        u_wrap = u - d;
        I2_block_addr_1 = I2_desc_shared + 16 * u_wrap;
        I2_block_addr_2 = I2_desc_shared + 16 * u_wrap + oneLine;
    //    result1 = computeMatchEnergy1(I1_block_addr_1, I2_block_addr_1, DESC_OFFSET_1);
    //    result2 = computeMatchEnergy1(I1_block_addr_1, I2_block_addr_1, DESC_OFFSET_2);
    //    result3 = computeMatchEnergy1(I1_block_addr_2, I2_block_addr_2, DESC_OFFSET_3);
        result4 = computeMatchEnergy1(I1_block_addr_2, I2_block_addr_2, DESC_OFFSET_4);
    //    sum = result1 + result2 + result3 + result4;
	sum = result4;
        if (sum<min_1_E) {
            min_2_E = min_1_E;
            min_2_d = min_1_d;
            min_1_E = sum;
            min_1_d = d;
        }
        else if (sum<min_2_E) {
            min_2_E = sum;
            min_2_d = d;
        }

    }

    if (min_1_d>=0 && min_2_d>=0 && (float)min_1_E < 0.85*(float)min_2_E)
      d1 =  min_1_d;

    sum = 0;
    if (d1 >= 0){
        min_1_E = MIN_1_E;
        min_1_d = MIN_1_D;
        min_2_E = MIN_2_E;
        min_2_d = MIN_2_D;
        u = u - d1;
        disp_max_valid = min(63, WIDTH - u - 5);
        I2_block_addr_1 = I2_desc_shared + 16 * u;
        I2_block_addr_2 = I2_desc_shared + 16 * u + 320 * 16;

        sum = 0;
    #pragma unroll
        for (int32_t i=0; i<16; i++)
          sum += abs((int32_t)(*(I2_line_addr+i))-128);
        if (sum<10){
            return;
        }
        sum = 0;

        for(int16_t d = 0; d <= disp_max_valid; d++){
            u_wrap = u + d;
            I1_block_addr_1 = I1_desc_shared + 16 * u_wrap;
            I1_block_addr_2 = I1_desc_shared + 16 * u_wrap + 320 * 16;
        //    result1 = computeMatchEnergy1(I1_block_addr_1, I2_block_addr_1, DESC_OFFSET_1);
        //    result2 = computeMatchEnergy1(I1_block_addr_1, I2_block_addr_1, DESC_OFFSET_2);
        //    result3 = computeMatchEnergy1(I1_block_addr_2, I2_block_addr_2, DESC_OFFSET_3);
            result4 = computeMatchEnergy1(I1_block_addr_2, I2_block_addr_2, DESC_OFFSET_4);

          //  sum = result1 + result2 + result3 + result4;
	sum = result4;
//            sum = computeMatchEnergy1_new(I2_desc_shared, I2_desc_shared + oneLine, I1_desc_shared, I1_desc_shared + oneLine, u, u_wrap);

            if (sum<min_1_E) {
                min_2_E = min_1_E;
                min_2_d = min_1_d;
                min_1_E = sum;
                min_1_d = d;
            }
            else if (sum<min_2_E) {
                min_2_E = sum;
                min_2_d = d;
            }
        }

        if (min_1_d>=0 && min_2_d>=0 && (float)min_1_E < 0.85*(float)min_2_E)
          d2 =  min_1_d;
        if( d2 >= 0 && abs(d2 - d1) <= 2 )
            D_can[x + y * D_can_width] = d1;
    }



}

void addCornerSupportPoints(vector<Elas::support_pt> &p_support, int32_t width, int32_t height) {

    // list of border points
    vector<Elas::support_pt> p_border;
    p_border.push_back(Elas::support_pt(0, 0, 0));
    p_border.push_back(Elas::support_pt(0, height - 1, 0));
    p_border.push_back(Elas::support_pt(width - 1, 0, 0));
    p_border.push_back(Elas::support_pt(width - 1, height - 1, 0));

    // find closest d
    for (int32_t i = 0; i<p_border.size(); i++) {
        int32_t best_dist = 10000000;
        for (int32_t j = 0; j<p_support.size(); j++) {
            int32_t du = p_border[i].u - p_support[j].u;
            int32_t dv = p_border[i].v - p_support[j].v;
            int32_t curr_dist = du*du + dv*dv;
            if (curr_dist<best_dist) {
                best_dist = curr_dist;
                p_border[i].d = p_support[j].d;
            }
        }
    }

    // for right image
    p_border.push_back(Elas::support_pt(p_border[2].u + p_border[2].d, p_border[2].v, p_border[2].d));
    p_border.push_back(Elas::support_pt(p_border[3].u + p_border[3].d, p_border[3].v, p_border[3].d));

    // add border points to support points
    for (int32_t i = 0; i<p_border.size(); i++)
        p_support.push_back(p_border[i]);
}


__global__ void removeInconsistentSupportPoints1(int16_t* D_can, int32_t D_can_width, int32_t D_can_height) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int u, v;

    if (x < D_can_width && y < D_can_height) {
        int16_t d_can = *(D_can + getAddressOffsetImage1(x, y, D_can_width));
        if (d_can >= 0) {
            int32_t support = 0;
            for (int32_t u_can_2 = x - INCON_WINDOW_SIZE; u_can_2 <= x + INCON_WINDOW_SIZE; u_can_2++) {
                for (int32_t v_can_2 = y - INCON_WINDOW_SIZE; v_can_2 <= y + INCON_WINDOW_SIZE; v_can_2++) {
                    if (u_can_2 >= 0 && v_can_2 >= 0 && u_can_2<D_can_width && v_can_2<D_can_height) {
                        int16_t d_can_2 = *(D_can + getAddressOffsetImage1(u_can_2, v_can_2, D_can_width));
                        if (d_can_2 >= 0 && abs(d_can - d_can_2) <= INCON_THRESHOLD)
                            support++;
                    }
                }
            }

            // invalidate support point if number of supporting points is too low
            if (support<INCON_MIN_SUPPORT)
                *(D_can + getAddressOffsetImage1(x, y, D_can_width)) = -1;
        }
    }
}

__global__ void removeRedundantSupportPoints1(int16_t* D_can, int32_t D_can_width, int32_t D_can_height,
    int32_t redun_max_dist, int32_t redun_threshold, bool vertical) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < D_can_width && y < D_can_height) {
        // parameters
        int32_t redun_dir_u[2] = { 0,0 };
        int32_t redun_dir_v[2] = { 0,0 };
        if (vertical) {
            redun_dir_v[0] = -1;
            redun_dir_v[1] = +1;
        }
        else {
            redun_dir_u[0] = -1;
            redun_dir_u[1] = +1;
        }
        int16_t d_can = *(D_can + getAddressOffsetImage1(x, y, D_can_width));
        if (d_can >= 0) {
            // check all directions for redundancy
            bool redundant = true;
            for (int32_t i = 0; i<2; i++) {

                // search for support
                int32_t u_can_2 = x;
                int32_t v_can_2 = y;
                int16_t d_can_2;
                bool support = false;
                for (int32_t j = 0; j<redun_max_dist; j++) {
                    u_can_2 += redun_dir_u[i];
                    v_can_2 += redun_dir_v[i];
                    if (u_can_2<0 || v_can_2<0 || u_can_2 >= D_can_width || v_can_2 >= D_can_height)
                        break;
                    d_can_2 = *(D_can + getAddressOffsetImage1(u_can_2, v_can_2, D_can_width));
                    if (d_can_2 >= 0 && abs(d_can - d_can_2) <= redun_threshold) {
                        support = true;
                        break;
                    }
                }

                // if we have no support => point is not redundant
                if (!support) {
                    redundant = false;
                    break;
                }
            }

            // invalidate support point if it is redundant
            if (redundant)
                *(D_can + getAddressOffsetImage1(x, y, D_can_width)) = -1;
        }
    }
}

vector<Elas::support_pt> computeSupportMatches_g(uint8_t* I_desc1, uint8_t* I_desc2, \
                                                 int8_t* D_sup_c, int8_t* D_sup_g)
{

    // create matrix for saving disparity candidates
    int32_t D_can_width = 60;  //[15,310] => 60
    int32_t D_can_height = 48; //[5, 230] => 46

    hipMemset(D_sup_g, -1, D_can_width*D_can_height * sizeof(int8_t));

    dim3 threads(BLOCKX, BLOCKY);
    dim3 grid(GRIDX, GRIDY);
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(sptMathKernel),hipFuncCachePreferShared);
    //compute support
    hipDeviceSynchronize();    //
    sptMathKernel << <grid, threads, 0, stream1>> > (D_can_width, D_can_height, D_sup_g, I_desc1, I_desc2);
    hipDeviceSynchronize();    //13ms

    //put D_sup to vector of support
    vector<Elas::support_pt> p_support;
    for (int32_t u_can = 0; u_can<D_can_width; u_can++)
        for (int32_t v_can = 0; v_can<D_can_height; v_can++)
            if (*(D_sup_c + u_can + v_can * D_can_width) >= 0)
                p_support.push_back(Elas::support_pt((u_can + 3)*D_candidate_stepsize,
                    (v_can + 1) * D_candidate_stepsize,
                    *(D_sup_c+ u_can + v_can * D_can_width)));


    return p_support;
}

/**
 * 3. over: compute support point
 * */


/**
 * 4. convert dis to cloud
 ***/

__global__ void Convert(float *D_g, float *cloud_g)
{
    int u = blockDim.x * blockIdx.x + threadIdx.x;
    int v = blockDim.y * blockIdx.y + threadIdx.y;
    v += 20;
    float w = 0, x = 0, y = 0, z = 0;
    float dis = D_g[u + v * WIDTH];


        w = 0.006669723997311648 * dis;
        x = (float)((u - 161.2100334167481) / w);
        y = (float)((219 - v - 119.9240913391113) / w); //has bug
        z = (float)(241.57918 / w);

    *(cloud_g + (u + v * WIDTH) * 3) = x;
    *(cloud_g + (u + v * WIDTH) * 3 + 1) = y;
    *(cloud_g + (u + v * WIDTH) * 3 + 2) = z;

//        *(cloud_g + (u + v * WIDTH) * 3)     = z;
//        *(cloud_g + (u + v * WIDTH) * 3 + 1) = y;
//        *(cloud_g + (u + v * WIDTH) * 3 + 2) = x;

    //            A = [u, v, d,1];
    //            Q = [1, 0, 0, -161.2100334167481; 0, 1, 0, -119.9240913391113;
    //                 0, 0, 0, 241.57918; 0, 0, 0.006669723997311648, 0]


}

int ConvertD2Z(float* D1_g,  float* cloud_g)
{
    dim3 threads(320, 1);
    dim3 grid(1, 200);
    Convert<<<grid, threads>>>(D1_g, cloud_g);
    hipDeviceSynchronize();
}


/**
 * 4. over: convert dis to cloud
 ***/



/****
 * 5. cuda_computeD
 ***/
int tri_size = 0;
__constant__ int32_t grid_dims_g[3] = {65, WIDTH/GRID_SIZE, HEIGH/GRID_SIZE} ;
__constant__ int8_t temp[] = {-14,-9,-2,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

__global__ void Triangle_Match1(Elas::triangle* tri, int32_t* disparity_grid,\
                                uint8_t* I1_desc, uint8_t* I2_desc, int8_t* P, \
                                int32_t plane_radius, bool right_image, float* D,  \
                                int8_t* tp, int tri_size)
{

    float plane_a = 0, plane_b = 0, plane_c = 0, plane_d = 0;

    int u = blockDim.x * blockIdx.x + threadIdx.x;
    int v = blockDim.y * blockIdx.y + threadIdx.y;
    int32_t id;
    __shared__ uint8_t      __I1_desc_share[320 * 16];
    __shared__ uint8_t      __I2_desc_share[320 * 16];

    for(int i = 0; i < 16; i += 1 )
    {
        __I1_desc_share[u + i*320] = I1_desc[v * 320*16 + u + i*320];
        __I2_desc_share[u + i*320] = I2_desc[v * 320*16 + u + i*320];
    }
    __syncthreads();

    id = tp[u + v * WIDTH];

        if(id > tri_size)
            printf("\nid out %d\n", id);

    plane_a = tri[id].t1a;
    plane_b = tri[id].t1b;
    plane_c = tri[id].t1c;
    plane_d = tri[id].t2a;


    bool valid = fabs(plane_a)<0.7 && fabs(plane_d)<0.7;
    const int32_t window_size = 2;
    // address of disparity we want to compute
    uint32_t d_addr;
    d_addr = getAddressOffsetImage1(u, v, WIDTH);

            uint8_t *I1_line_addr, *I2_line_addr;
            I2_line_addr = __I2_desc_share ;
            uint8_t* I1_block_addr = __I1_desc_share + 16 * u;

            // does this patch have enough texture?
            int32_t sum = 0;
//int32_t match_texture = 1;
//        //#pragma unroll
//            for (int32_t i = 0; i<16; i++)
//                sum += abs((int32_t)(*(I1_block_addr + i)) - 127);
//            if (sum<match_texture)
//                return;

            // compute disparity, min disparity and max disparity of plane prior
//            int32_t d_plane = (int32_t)(plane_a*(float)u + plane_b*(float)v + plane_c);
            int32_t d_plane = (int32_t)(0);
            int32_t d_plane_min = max(d_plane - plane_radius, 0);
            int32_t d_plane_max = min(d_plane + plane_radius, grid_dims_g[0] - 2);

            // get grid pointer
            int32_t  grid_x = (int32_t)floor((float)u / (float)GRID_SIZE);
            int32_t  grid_y = (int32_t)floor((float)v / (float)GRID_SIZE);

            uint32_t grid_addr = getAddressOffsetGrid1(grid_x, grid_y, 0, grid_dims_g[1], grid_dims_g[0]);
            int32_t  num_grid = *(disparity_grid + grid_addr);
            int32_t* d_grid = disparity_grid + grid_addr + 1;

            // loop variables
            int32_t d_curr, u_warp, val;
            int32_t min_val = 10000;
            int32_t min_d = -1;


            // left image

            if (!right_image) {
        #pragma unroll
                for (int32_t i = 0; i<num_grid; i++) {
                    d_curr = d_grid[i];
                    if (d_curr<d_plane_min || d_curr>d_plane_max) {
                        u_warp = u - d_curr;
                       updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);
                    }
                }
        #pragma unroll
                for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
                    u_warp = u - d_curr;
//                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(temp + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                }
            }
            else {
        #pragma unroll
                for (int32_t i = 0; i<num_grid; i++) {
                    d_curr = d_grid[i];
                    if (d_curr<d_plane_min || d_curr>d_plane_max) {
                        u_warp = u + d_curr;
                        if (u_warp<window_size || u_warp >= WIDTH - window_size)
                            continue;
                        updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);
                    }
                }
        #pragma unroll
                for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
                    u_warp = u + d_curr;
                    if (u_warp<window_size || u_warp >= WIDTH - window_size)
                        continue;
//                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(temp + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                }
            }



            // set disparity value
//            if (min_d >= 0){
            if (min_d > 0){
                *(D + d_addr) = min_d; // MAP value (min neg-Log probability)
            }else          *(D + d_addr) = -1;    // invalid disparity
}

void computeTrianglePoints(const vector<Elas::support_pt> &p_support, const vector<Elas::triangle> &tri, \
                           bool right_image, int32_t width, int32_t TRI_SIZE, int8_t* tp) {

    // loop variables
    int32_t c1, c2, c3;
//    float plane_a, plane_b, plane_c, plane_d;

    // for all triangles do
    for (uint32_t i = 0; i<TRI_SIZE; i++) {
        int num = 0;
        // get plane parameters
        uint32_t p_i = i * 3;

        // triangle corners
        c1 = tri[i].c1;
        c2 = tri[i].c2;
        c3 = tri[i].c3;

        // sort triangle corners wrt. u (ascending)
        float tri_u[3];
        if (!right_image) {     //左图像
            tri_u[0] = p_support[c1].u;
            tri_u[1] = p_support[c2].u;
            tri_u[2] = p_support[c3].u;
        }
        else {                //右图像
            tri_u[0] = p_support[c1].u - p_support[c1].d;
            tri_u[1] = p_support[c2].u - p_support[c2].d;
            tri_u[2] = p_support[c3].u - p_support[c3].d;
        }
        float tri_v[3] = { p_support[c1].v,p_support[c2].v,p_support[c3].v };

        for (uint32_t j = 0; j<3; j++) {
            for (uint32_t k = 0; k<j; k++) {
                if (tri_u[k]>tri_u[j]) {
                    float tri_u_temp = tri_u[j]; tri_u[j] = tri_u[k]; tri_u[k] = tri_u_temp;
                    float tri_v_temp = tri_v[j]; tri_v[j] = tri_v[k]; tri_v[k] = tri_v_temp;
                }
            }
        }

        // rename corners
        float A_u = tri_u[0]; float A_v = tri_v[0];
        float B_u = tri_u[1]; float B_v = tri_v[1];
        float C_u = tri_u[2]; float C_v = tri_v[2];

        // compute straight lines connecting triangle corners
        float AB_a = 0; float AC_a = 0; float BC_a = 0;
        if ((int32_t)(A_u) != (int32_t)(B_u)) AB_a = (A_v - B_v) / (A_u - B_u);
        if ((int32_t)(A_u) != (int32_t)(C_u)) AC_a = (A_v - C_v) / (A_u - C_u);
        if ((int32_t)(B_u) != (int32_t)(C_u)) BC_a = (B_v - C_v) / (B_u - C_u);
        float AB_b = A_v - AB_a*A_u;
        float AC_b = A_v - AC_a*A_u;
        float BC_b = B_v - BC_a*B_u;


        // first part (triangle corner A->B)
        if ((int32_t)(A_u) != (int32_t)(B_u)) {
            for (int32_t u = max((int32_t)A_u, 0); u < min((int32_t)B_u, width); u++) {
                if (!param.subsampling || u % 2 == 0) {
                    int32_t v_1 = (uint32_t)(AC_a*(float)u + AC_b);
                    int32_t v_2 = (uint32_t)(AB_a*(float)u + AB_b);
                    for (int32_t v = min(v_1, v_2); v < max(v_1, v_2); v++)
                        if (!param.subsampling || v % 2 == 0)
                        {
//                            *((int16_t*)(tp + 2 * u + v * 2 * width)) = u;
//                            *((int16_t*)(tp + 2 * u + v * 2 * width) + 1) = v;
//                            *(tp + 2 * u + v * 2 * width + 1) = i;
                            *(tp + u + v * width ) = i;
//                            num++;
                        }
                }
            }

        }

        // second part (triangle corner B->C)
        if ((int32_t)(B_u) != (int32_t)(C_u)) {
            for (int32_t u = max((int32_t)B_u, 0); u < min((int32_t)C_u, width); u++) {
                if (!param.subsampling || u % 2 == 0) {
                    int32_t v_1 = (uint32_t)(AC_a*(float)u + AC_b);
                    int32_t v_2 = (uint32_t)(BC_a*(float)u + BC_b);
                    for (int32_t v = min(v_1, v_2); v < max(v_1, v_2); v++)
                        if (!param.subsampling || v % 2 == 0)
                        {
//                            *((int16_t*)(tp + 2 * u + v * 2 * width)) = u;
//                            *((int16_t*)(tp + 2 * u + v * 2 * width) + 1) = v;
//                            *(tp + 2 * u + v * 2 * width + 1) = i;
                              *(tp + u + v * width) = i;
//                            num++;
                        }
                }
            }
        }
//        tri[i].pointNum = num;
    }

}

hipError_t err;

int32_t dims[3] = {WIDTH, HEIGH, WIDTH};

void cuda_computeD(int32_t* disparity_grid_1, int32_t* disparity_grid_2,  vector<Elas::support_pt> &p_support, \
              vector<Elas::triangle> &tri_1, vector<Elas::triangle> &tri_2, \
              float* D1, float* D2, uint8_t* I1, uint8_t* I2, int8_t* P_g,\
             int8_t *tp1_g, int8_t* tp2_g, int8_t* tp1_c, int8_t* tp2_c)
{    
    int32_t width, height, bpl;
    clock_t t1, t2;

    // get width, height and bytes per line
    width = dims[0];    //
    height = dims[1];
    bpl = dims[2];  //

    // allocate memory for disparity grid
    int32_t grid_width = 16; //(int32_t)ceil((float)width / (float)20);
    int32_t grid_height = 12; //(int32_t)ceil((float)height / (float)20);
    int32_t grid_dims[3] = { 63 + 2,grid_width,grid_height };



    int32_t P_SUPPORT_SIZE = p_support.size();
    int32_t TRI_SIZE1 = tri_1.size();
    int32_t TRI_SIZE2 = tri_2.size();
    tri_size = TRI_SIZE1;

    int8_t* tp1_cpu = tp1_c;
    int8_t* tp2_cpu = tp2_c;
    int8_t *tp1_gpu = tp1_g;
    int8_t *tp2_gpu = tp2_g;


    computeTrianglePoints(p_support, tri_1, 0, width, TRI_SIZE1, tp1_cpu);
    computeTrianglePoints(p_support, tri_2, 1, width, TRI_SIZE2, tp2_cpu);

    Elas::triangle* tri_gpu_1, *tri_gpu_2;
    int32_t *P_gpu = NULL;  //action::::  cannot delete;

    hipMalloc((void **)&tri_gpu_1, sizeof(Elas::triangle) * TRI_SIZE1);
    hipMalloc((void **)&tri_gpu_2, sizeof(Elas::triangle) * TRI_SIZE2);
    hipMalloc((void **)&P_gpu, sizeof(int8_t) * 64);
    err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));


    hipMemcpy(tri_gpu_1, &tri_1[0], sizeof(Elas::triangle) * TRI_SIZE1, hipMemcpyHostToDevice);
    hipMemcpy(tri_gpu_2, &tri_2[0], sizeof(Elas::triangle) * TRI_SIZE2, hipMemcpyHostToDevice);
    hipMemcpy(P_gpu, P_g, sizeof(int8_t) * 64, hipMemcpyHostToDevice);
    err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));

    int32_t plane_radius = 2; //(int32_t)max((float)ceil(param.sigma*param.sradius), (float)2.0);

    dim3 threads(320, 1);
    dim3 grid(1, 240);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if(0 != err) printf("cuda error: %s\n", hipGetErrorString(err));

    printf("goin Triangle_match kernel\n");
    Triangle_Match1 << <grid, threads, 0>> > (tri_gpu_1, disparity_grid_1, \
                  I1, I2, P_g, plane_radius, 0, D1, tp1_gpu, tri_size);

    tri_size = TRI_SIZE2;
    Triangle_Match1 << <grid, threads, 0>> > (tri_gpu_2, disparity_grid_2, \
                  I2, I1, P_g, plane_radius, 1, D2, tp2_gpu, tri_size);
    err = hipGetLastError();
    if(0 != err) printf("Triangle_Match1 cuda error: %s\n", hipGetErrorString(err));

    hipDeviceSynchronize();
    err = hipGetLastError();
    if(0 != err) printf("Triangle_Match1 cuda error: %s\n", hipGetErrorString(err));

    hipFree((void*)tri_gpu_1);
    hipFree((void*)tri_gpu_2);
    hipFree((void*)P_gpu);
    //  hipDeviceSynchronize();



}


/****
 * 5. over: cuda_computeD
 ***/
