#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

#include "cv.h"
#include "highgui.h"
#include "elas.h"
#include <vector>
#include "triangle.h"
#include "matrix.h"
#include <stdlib.h>

using namespace std;

enum setting { ROBOTICS, MIDDLEBURY };

// parameter settings
struct parameters {
    int32_t disp_min;               // min disparity
    int32_t disp_max;               // max disparity
    float   support_threshold;      // max. uniqueness ratio (best vs. second best support match)  最大视差唯一性百分比
    int32_t support_texture;        // min texture for support points      最小纹理支持点
    int32_t candidate_stepsize;     // step size of regular grid on which support points are matched
    int32_t incon_window_size;      // window size of inconsistent support point check
    int32_t incon_threshold;        // disparity similarity threshold for support point to be considered consistent
    int32_t incon_min_support;      // minimum number of consistent support points
    bool    add_corners;            // add support points at image corners with nearest neighbor disparities
    int32_t grid_size;              // size of neighborhood for additional support point extrapolation
    float   beta;                   // image likelihood parameter
    float   gamma;                  // prior constant
    float   sigma;                  // prior sigma
    float   sradius;                // prior sigma radius
    int32_t match_texture;          // min texture for dense matching
    int32_t lr_threshold;           // disparity threshold for left/right consistency check
    float   speckle_sim_threshold;  // similarity threshold for speckle segmentation
    int32_t speckle_size;           // maximal size of a speckle (small speckles get removed)
    int32_t ipol_gap_width;         // interpolate small gaps (left<->right, top<->bottom)
    bool    filter_median;          // optional median filter (approximated)
    bool    filter_adaptive_mean;   // optional adaptive mean filter (approximated)
    bool    postprocess_only_left;  // saves time by not postprocessing the right image
    bool    subsampling;            // saves time by only computing disparities for each 2nd pixel
                                    // note: for this option D1 and D2 must be passed with size
                                    //       width/2 x height/2 (rounded towards zero)

                                    // constructor
    parameters(setting s = ROBOTICS) {

        // default settings in a robotics environment
        // (do not produce results in half-occluded areas
        //  and are a bit more robust towards lighting etc.)  //默认设置为实验环境，不能在half-occluded环境和大量光照条件下使用
        if (s == ROBOTICS) {
            disp_min = 0;
            disp_max = 63;
            support_threshold = 0.85;
            support_texture = 10;
            candidate_stepsize = 5;
            incon_window_size = 5;
            incon_threshold = 5;
            incon_min_support = 5;
            add_corners = 0;
            grid_size = 20;
            beta = 0.02;
            gamma = 3;
            sigma = 1;
            sradius = 2;
            match_texture = 1;                //dense matching的最小纹理
            lr_threshold = 2;                //一致性检测阈值
            speckle_sim_threshold = 2;                //删除细小片段相似性分割阈值
            speckle_size = 200;              //删除细小片段size
            ipol_gap_width = 7;                //间隙插值阈值
            filter_median = 0;
            filter_adaptive_mean = 0;
            postprocess_only_left = 1;
            subsampling = 0;



            // default settings for middlebury benchmark
            // (interpolate all missing disparities)   middlebury基准，插入所有失踪的视差
        }
        else {
            disp_min = 0;
            disp_max = 63;
            support_threshold = 0.85;
            support_texture = 10;
            candidate_stepsize = 5;
            incon_window_size = 5;
            incon_threshold = 5;
            incon_min_support = 5;
            add_corners = 1;
            grid_size = 20;
            beta = 0.02;
            gamma = 5;
            sigma = 1;
            sradius = 3;
            match_texture = 0;
            lr_threshold = 2;
            speckle_sim_threshold = 1;
            speckle_size = 200;
            ipol_gap_width = 5000;
            filter_median = 1;
            filter_adaptive_mean = 0;
            postprocess_only_left = 0;
            subsampling = 0;
        }
    }
};

// parameter set
parameters param(ROBOTICS);

static hipStream_t stream1, stream2, stream3, stream4;

struct support_pt {
    int32_t u;
    int32_t v;
    int32_t d;
    support_pt(int32_t u, int32_t v, int32_t d) :u(u), v(v), d(d) {}
};

struct support_pt1 {
    int32_t u;
    int32_t v;
    int32_t d;
};

struct triangle {
    int32_t c1, c2, c3;
    float   t1a, t1b, t1c;
    float   t2a, t2b, t2c;
    triangle(int32_t c1, int32_t c2, int32_t c3) :c1(c1), c2(c2), c3(c3) {}
};


struct triangle1 {
    int32_t c1, c2, c3;
    float   t1a, t1b, t1c;
    float   t2a, t2b, t2c;
    int32_t pointNum;
};


__device__ uint32_t getAddressOffsetImage1(const int32_t& u, const int32_t& v, const int32_t& width) {
    return v*width + u;
}

inline uint32_t getAddressOffsetImage(const int32_t& u, const int32_t& v, const int32_t& width) {
    return v*width + u;
}



__device__ unsigned int computeMatchEnergy1(unsigned char* dst1, unsigned char* dst2, int offset) {
    unsigned int a, b, c, e, r0, r4;

    a = abs(*(dst1 + offset) - *(dst2 + offset)) + abs(*(dst1 + offset + 1) - *(dst2 + offset + 1));
    b = abs(*(dst1 + offset + 2) - *(dst2 + offset + 2)) + abs(*(dst1 + offset + 3) - *(dst2 + offset + 3));
    c = abs(*(dst1 + offset + 4) - *(dst2 + offset + 4)) + abs(*(dst1 + offset + 5) - *(dst2 + offset + 5));
    e = abs(*(dst1 + offset + 6) - *(dst2 + offset + 6)) + abs(*(dst1 + offset + 7) - *(dst2 + offset + 7));
    r0 = a + b + c + e;

    a = abs(*(dst1 + offset + 8) - *(dst2 + offset + 8)) + abs(*(dst1 + offset + 9) - *(dst2 + offset + 9));
    b = abs(*(dst1 + offset + 10) - *(dst2 + offset + 10)) + abs(*(dst1 + offset + 11) - *(dst2 + offset + 11));
    c = abs(*(dst1 + offset + 12) - *(dst2 + offset + 12)) + abs(*(dst1 + offset + 13) - *(dst2 + offset + 13));
    e = abs(*(dst1 + offset + 14) - *(dst2 + offset + 14)) + abs(*(dst1 + offset + 15) - *(dst2 + offset + 15));
    r4 = a + b + c + e;

    return r0 + r4;
}

inline uint32_t getAddressOffsetGrid(const int32_t& x, const int32_t& y, const int32_t& d, const int32_t& width, const int32_t& disp_num) {
    return (y*width + x)*disp_num + d;
}

__device__ uint32_t getAddressOffsetGrid1(const int32_t& x, const int32_t& y, const int32_t& d, const int32_t& width, const int32_t& disp_num) {
    return (y*width + x)*disp_num + d;
}


__device__ void updatePosteriorMinimumNew(unsigned char* dst1, unsigned char* dst2, const int32_t &d, int32_t &val, int32_t &min_val, int32_t &min_d) {
    val = computeMatchEnergy1(dst1, dst2, 0);
    if (val<min_val) {
        min_val = val;
        min_d = d;
    }
}

__device__ void updatePosteriorMinimumNew1(unsigned char* dst1, unsigned char* dst2, const int32_t &d, const int32_t &w, int32_t &val, int32_t &min_val, int32_t &min_d) {
    val = computeMatchEnergy1(dst1, dst2, 0) + w;
    if (val<min_val) {
        min_val = val;
        min_d = d;
    }
}

int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}



__device__ void findMatch(int32_t &u, int32_t &v, float &plane_a, float &plane_b, float &plane_c, int32_t* disparity_grid, int32_t *grid_dims,
    uint8_t* I1_desc, uint8_t* I2_desc, int32_t *P, int32_t &plane_radius, bool &valid, bool &right_image, float* D,
    int32_t width, int32_t height, bool subsampling, int32_t match_texture, int32_t grid_size) {

    // get image width and height
    const int32_t disp_num = grid_dims[0] - 1;
    const int32_t window_size = 2;

    // address of disparity we want to compute
    uint32_t d_addr;
    if (subsampling) d_addr = getAddressOffsetImage1(u / 2, v / 2, width / 2);
    else                   d_addr = getAddressOffsetImage1(u, v, width);




    // compute line start address
    int32_t  line_offset = 16 * width*max(min(v, height - 3), 2);
    uint8_t *I1_line_addr, *I2_line_addr;
    if (!right_image) {
        I1_line_addr = I1_desc + line_offset;
        I2_line_addr = I2_desc + line_offset;
    }
    else {
        I1_line_addr = I2_desc + line_offset;
        I2_line_addr = I1_desc + line_offset;
    }

    // compute I1 block start address
    uint8_t* I1_block_addr = I1_line_addr + 16 * u;

    // does this patch have enough texture?
    int32_t sum = 0;

//#pragma unroll
//    for (int32_t i = 0; i<16; i++)
//        sum += abs((int32_t)(*(I1_block_addr + i)) - 127);
//    if (sum<match_texture)
//        return;

    // compute disparity, min disparity and max disparity of plane prior
    int32_t d_plane = (int32_t)(plane_a*(float)u + plane_b*(float)v + plane_c);
    int32_t d_plane_min = max(d_plane - plane_radius, 0);
    int32_t d_plane_max = min(d_plane + plane_radius, disp_num - 1);

    // get grid pointer
    int32_t  grid_x = (int32_t)floor((float)u / (float)grid_size);
    int32_t  grid_y = (int32_t)floor((float)v / (float)grid_size);
    uint32_t grid_addr = getAddressOffsetGrid1(grid_x, grid_y, 0, grid_dims[1], grid_dims[0]);
    int32_t  num_grid = *(disparity_grid + grid_addr);



    int32_t* d_grid = disparity_grid + grid_addr + 1;

    // loop variables
    int32_t d_curr, u_warp, val;
    int32_t min_val = 10000;
    int32_t min_d = -1;


    // left image

    if (!right_image) {
//#pragma unroll
        for (int32_t i = 0; i<num_grid; i++) {
            d_curr = d_grid[i];
            if (d_curr<d_plane_min || d_curr>d_plane_max) {
                u_warp = u - d_curr;
                if (u_warp<window_size || u_warp >= width - window_size)
                    continue;
                //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
                updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);

            }
        }
//#pragma unroll
        for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
            u_warp = u - d_curr;
            if (u_warp<window_size || u_warp >= width - window_size)
                continue;
            //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
            updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
        }

        // right image
    }
    else {
//#pragma unroll
        for (int32_t i = 0; i<num_grid; i++) {
            d_curr = d_grid[i];
            if (d_curr<d_plane_min || d_curr>d_plane_max) {
                u_warp = u + d_curr;
                if (u_warp<window_size || u_warp >= width - window_size)
                    continue;
                //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
                updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);
            }
        }
//#pragma unroll
        for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
            u_warp = u + d_curr;
            if (u_warp<window_size || u_warp >= width - window_size)
                continue;
            //updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
            updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
        }
    }



    // set disparity value
    if (min_d >= 0) *(D + d_addr) = min_d; // MAP value (min neg-Log probability)
    else          *(D + d_addr) = -1;    // invalid disparity

}


//__constant__ int32_t grid_dims_g[3];
//__constant__ int32_t P_g[grid_dims_g[0]-1];


__global__ void Triangle_Match1(triangle1* tri, int32_t* disparity_grid, int32_t *grid_dims,\
                                uint8_t* I1_desc, uint8_t* I2_desc, int32_t* P, \
                                int32_t plane_radius, bool right_image, float* D, int32_t width, int32_t height, \
                                int32_t TRI_SIZE, bool subsampling, int32_t match_texture, int32_t grid_size,\
                                 int32_t* tp)
{




//     __syncthreads();

    float plane_a, plane_b, plane_c, plane_d;
    int32_t c1, c2, c3;
        int u = blockDim.x * blockIdx.x + threadIdx.x;
        int v = blockDim.y * blockIdx.y + threadIdx.y;
    int id;
        __shared__ uint8_t I2_desc_share[320 * 16];
        __shared__ int32_t P_g[64];
        __shared__ uint8_t I1_desc_share[320 * 16];
//        __shared__ float plane_a_g[320], plane_b_g[320], plane_c_g[320],plane_d_g[320];

        for(int i = 0; i < 16; i++ )
        {
            I1_desc_share[u*16 + i] = I1_desc[v * 320*16 + u*16 + i];
        }

        for(int i = 0; i < 16; i++ )
        {
            I2_desc_share[u*16 + i] = I2_desc[v * 320 + u*16 + i];
        }

    __syncthreads();

//    if ( (u > 2) && (u < width - 2)  && (v < height) ) {
        if (tp[2 * u + v * 2 * width] != -1) {

            id = tp[2 * u + v * 2 * width + 1];

            // get plane parameters
            if (!right_image) {    //左图像
                plane_a = tri[id].t1a;
                plane_b = tri[id].t1b;
                plane_c = tri[id].t1c;
                plane_d = tri[id].t2a;
            }
            else {               //右图像
                plane_a = tri[id].t2a;
                plane_b = tri[id].t2b;
                plane_c = tri[id].t2c;
                plane_d = tri[id].t1a;
            }

            bool valid = fabs(plane_a)<0.7 && fabs(plane_d)<0.7;
//            findMatch(x, y, plane_a, plane_b, plane_c, disparity_grid, grid_dims,
//                I1_desc, I2_desc, P, plane_radius, valid, right_image, D, width,\
//                      height, subsampling, match_texture, grid_size);

            // get image width and height
            const int32_t disp_num = grid_dims[0] - 1;
            const int32_t window_size = 2;

            // address of disparity we want to compute
            uint32_t d_addr;
            d_addr = getAddressOffsetImage1(u, v, width);




            // compute line start address
            int32_t  line_offset = 16 * width*max(min(v, height - 3), 2);
            uint8_t *I1_line_addr, *I2_line_addr;
//            if (!right_image) {
                I1_line_addr = I1_desc + line_offset;
                I2_line_addr = I2_desc + line_offset;
//            }
//            else {
//                I1_line_addr = I2_desc + line_offset;
//                I2_line_addr = I1_desc + line_offset;
//            }

            // compute I1 block start address
//            uint8_t* I1_block_addr = I1_line_addr + 16 * u;
uint8_t* I1_block_addr = I1_desc_share + 16 * u;
            // does this patch have enough texture?
            int32_t sum = 0;

        //#pragma unroll
        //    for (int32_t i = 0; i<16; i++)
        //        sum += abs((int32_t)(*(I1_block_addr + i)) - 127);
        //    if (sum<match_texture)
        //        return;

            // compute disparity, min disparity and max disparity of plane prior
            int32_t d_plane = (int32_t)(plane_a*(float)u + plane_b*(float)v + plane_c);
            int32_t d_plane_min = max(d_plane - plane_radius, 0);
            int32_t d_plane_max = min(d_plane + plane_radius, disp_num - 1);

            // get grid pointer
            int32_t  grid_x = (int32_t)floor((float)u / (float)grid_size);
            int32_t  grid_y = (int32_t)floor((float)v / (float)grid_size);
            uint32_t grid_addr = getAddressOffsetGrid1(grid_x, grid_y, 0, grid_dims[1], grid_dims[0]);
            int32_t  num_grid = *(disparity_grid + grid_addr);



            int32_t* d_grid = disparity_grid + grid_addr + 1;

            // loop variables
            int32_t d_curr, u_warp, val;
            int32_t min_val = 10000;
            int32_t min_d = -1;


            // left image

            if (!right_image) {
        //#pragma unroll
                for (int32_t i = 0; i<num_grid; i++) {
                    d_curr = d_grid[i];
                    if (d_curr<d_plane_min || d_curr>d_plane_max) {
                        u_warp = u - d_curr;
                        if (u_warp<window_size || u_warp >= width - window_size)
                            continue;
//                        updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
                        updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);
                          updatePosteriorMinimumNew(I1_block_addr, I2_desc_share + 16 * u_warp, d_curr, val, min_val, min_d);

                    }
                }
        //#pragma unroll
                for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
                    u_warp = u - d_curr;
                    if (u_warp<window_size || u_warp >= width - window_size)
                        continue;
//                    updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
//                    updatePosteriorMinimumNew1(I1_block_addr, I2_desc_share + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                }

                // right image
            }
            else {
        //#pragma unroll
                for (int32_t i = 0; i<num_grid; i++) {
                    d_curr = d_grid[i];
                    if (d_curr<d_plane_min || d_curr>d_plane_max) {
                        u_warp = u + d_curr;
                        if (u_warp<window_size || u_warp >= width - window_size)
                            continue;
//                        updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,xmm1,xmm2,val,min_val,min_d);
                        updatePosteriorMinimumNew(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, val, min_val, min_d);
//                        updatePosteriorMinimumNew(I1_block_addr, I2_desc_share + 16 * u_warp, d_curr, val, min_val, min_d);
                    }
                }
        //#pragma unroll
                for (d_curr = d_plane_min; d_curr <= d_plane_max; d_curr++) {
                    u_warp = u + d_curr;
                    if (u_warp<window_size || u_warp >= width - window_size)
                        continue;
//                    updatePosteriorMinimum((__m128i*)(I2_line_addr+16*u_warp),d_curr,valid?*(P+abs(d_curr-d_plane)):0,xmm1,xmm2,val,min_val,min_d);
                    updatePosteriorMinimumNew1(I1_block_addr, I2_line_addr + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
//                    updatePosteriorMinimumNew1(I1_block_addr, I2_desc_share + 16 * u_warp, d_curr, valid ? *(P + abs(d_curr - d_plane)) : 0, val, min_val, min_d);
                }
            }



            // set disparity value
            if (min_d >= 0) *(D + d_addr) = min_d; // MAP value (min neg-Log probability)
            else          *(D + d_addr) = -1;    // invalid disparity





//        }
    }
}



//void computeTrianglePoints(support_pt1* p_support, triangle1* tri, bool right_image, int32_t width, int32_t TRI_SIZE, int32_t* tp) {
void computeTrianglePoints(const vector<Elas::support_pt> &p_support, const vector<Elas::triangle> &tri, \
                           bool right_image, int32_t width, int32_t TRI_SIZE, int32_t* tp) {

    // loop variables
    int32_t c1, c2, c3;
    float plane_a, plane_b, plane_c, plane_d;

    // for all triangles do
    for (uint32_t i = 0; i<TRI_SIZE; i++) {
        int num = 0;
        // get plane parameters
        uint32_t p_i = i * 3;
        if (!right_image) {    //左图像
            plane_a = tri[i].t1a;
            plane_b = tri[i].t1b;
            plane_c = tri[i].t1c;
            plane_d = tri[i].t2a;
        }
        else {               //右图像
            plane_a = tri[i].t2a;
            plane_b = tri[i].t2b;
            plane_c = tri[i].t2c;
            plane_d = tri[i].t1a;
        }

        // triangle corners
        c1 = tri[i].c1;
        c2 = tri[i].c2;
        c3 = tri[i].c3;

        // sort triangle corners wrt. u (ascending)
        float tri_u[3];
        if (!right_image) {     //左图像
            tri_u[0] = p_support[c1].u;
            tri_u[1] = p_support[c2].u;
            tri_u[2] = p_support[c3].u;
        }
        else {                //右图像
            tri_u[0] = p_support[c1].u - p_support[c1].d;
            tri_u[1] = p_support[c2].u - p_support[c2].d;
            tri_u[2] = p_support[c3].u - p_support[c3].d;
        }
        float tri_v[3] = { p_support[c1].v,p_support[c2].v,p_support[c3].v };

        for (uint32_t j = 0; j<3; j++) {
            for (uint32_t k = 0; k<j; k++) {
                if (tri_u[k]>tri_u[j]) {
                    float tri_u_temp = tri_u[j]; tri_u[j] = tri_u[k]; tri_u[k] = tri_u_temp;
                    float tri_v_temp = tri_v[j]; tri_v[j] = tri_v[k]; tri_v[k] = tri_v_temp;
                }
            }
        }

        // rename corners
        float A_u = tri_u[0]; float A_v = tri_v[0];
        float B_u = tri_u[1]; float B_v = tri_v[1];
        float C_u = tri_u[2]; float C_v = tri_v[2];

        // compute straight lines connecting triangle corners
        float AB_a = 0; float AC_a = 0; float BC_a = 0;
        if ((int32_t)(A_u) != (int32_t)(B_u)) AB_a = (A_v - B_v) / (A_u - B_u);
        if ((int32_t)(A_u) != (int32_t)(C_u)) AC_a = (A_v - C_v) / (A_u - C_u);
        if ((int32_t)(B_u) != (int32_t)(C_u)) BC_a = (B_v - C_v) / (B_u - C_u);
        float AB_b = A_v - AB_a*A_u;
        float AC_b = A_v - AC_a*A_u;
        float BC_b = B_v - BC_a*B_u;


        // first part (triangle corner A->B)
        if ((int32_t)(A_u) != (int32_t)(B_u)) {
            for (int32_t u = max((int32_t)A_u, 0); u < min((int32_t)B_u, width); u++) {
                if (!param.subsampling || u % 2 == 0) {
                    int32_t v_1 = (uint32_t)(AC_a*(float)u + AC_b);
                    int32_t v_2 = (uint32_t)(AB_a*(float)u + AB_b);
                    for (int32_t v = min(v_1, v_2); v < max(v_1, v_2); v++)
                        if (!param.subsampling || v % 2 == 0)
                        {
                            *((int16_t*)(tp + 2 * u + v * 2 * width)) = u;
                            *((int16_t*)(tp + 2 * u + v * 2 * width) + 1) = v;
                            *(tp + 2 * u + v * 2 * width + 1) = i;
                            num++;
                        }
                }
            }

        }

        // second part (triangle corner B->C)
        if ((int32_t)(B_u) != (int32_t)(C_u)) {
            for (int32_t u = max((int32_t)B_u, 0); u < min((int32_t)C_u, width); u++) {
                if (!param.subsampling || u % 2 == 0) {
                    int32_t v_1 = (uint32_t)(AC_a*(float)u + AC_b);
                    int32_t v_2 = (uint32_t)(BC_a*(float)u + BC_b);
                    for (int32_t v = min(v_1, v_2); v < max(v_1, v_2); v++)
                        if (!param.subsampling || v % 2 == 0)
                        {
                            *((int16_t*)(tp + 2 * u + v * 2 * width)) = u;
                            *((int16_t*)(tp + 2 * u + v * 2 * width) + 1) = v;
                            *(tp + 2 * u + v * 2 * width + 1) = i;
                            num++;
                        }
                }
            }
        }
//        tri[i].pointNum = num;
    }

}



#define WEIGH 320
#define HEIGH 240




void cuda_computeD(int32_t* disparity_grid_1, int32_t* disparity_grid_2,  vector<Elas::support_pt> &p_support, \
              vector<Elas::triangle> &tri_1, vector<Elas::triangle> &tri_2, \
              float* D1, float* D2, uint8_t* I1, uint8_t* I2, int dim)
{



    clock_t t1, t2;

    int32_t width, height, bpl;

    uint8_t* I_desc1 = NULL;
    uint8_t* I_desc2 = NULL;
    int32_t* grid_dims_gpu = NULL;
    int32_t* disparity_grid_gpu_1 = NULL;
    int32_t* disparity_grid_gpu_2 = NULL;
    float* D1_gpu = NULL;
    float* D2_gpu = NULL;
    int32_t* P_gpu = NULL;
    triangle1* tri_gpu_1, *tri_gpu_2;

hipError_t err;

    int32_t dims[3] = {WEIGH,HEIGH,WEIGH};

    // get width, height and bytes per line
    width = dims[0];    //715*492
    height = dims[1];
    bpl = width + 15 - (width - 1) % 16;  //720

    // allocate memory for disparity grid
    int32_t grid_width = (int32_t)ceil((float)width / (float)20);
    int32_t grid_height = (int32_t)ceil((float)height / (float)20);
    int32_t grid_dims[3] = { 63 + 2,grid_width,grid_height };
//    grid_dims[3] = { 63 + 2,grid_width,grid_height };




int32_t P_SUPPORT_SIZE = p_support.size();
int32_t TRI_SIZE1 = tri_1.size();
int32_t TRI_SIZE2 = tri_2.size();

int32_t* tp1_cpu, *tp2_cpu;
int32_t *tp1_gpu, *tp2_gpu;


cout<<"P_SUPPORT_SIZE: "<<P_SUPPORT_SIZE<<endl;
cout<< "TRI_SIZE1: " << TRI_SIZE1 <<endl;
cout<< "TRI_SIZE2: " << TRI_SIZE2 <<endl;

tp2_cpu = (int32_t*)malloc(sizeof(int32_t) * width * height * 2);
tp1_cpu = (int32_t*)malloc(sizeof(int32_t) * width * height * 2);


for (int i = 0; i < width * 2; i++) {
    for (int j = 0; j < height; j++) {
        tp1_cpu[i + j * width * 2] = -1;
        tp2_cpu[i + j * width * 2] = -1;
    }
}

t1 = clock();

//computeTrianglePoints(p_support_cpu, tri_cpu_1, 0, width, TRI_SIZE1, tp1_cpu);
//computeTrianglePoints(p_support_cpu, tri_cpu_2, 1, width, TRI_SIZE2, tp2_cpu);

computeTrianglePoints(p_support, tri_1, 0, width, TRI_SIZE1, tp1_cpu);
computeTrianglePoints(p_support, tri_2, 1, width, TRI_SIZE2, tp2_cpu);

t2 = clock();

printf("computeTripoints : %ldms\n", (t2 - t1)/1000);

hipMalloc((void **)&tp1_gpu, sizeof(int32_t) * width * height * 2);
hipMalloc((void **)&tp2_gpu, sizeof(int32_t) * width * height * 2);
hipMemcpy(tp1_gpu, tp1_cpu, sizeof(int32_t) * width * height * 2, hipMemcpyHostToDevice);
hipMemcpy(tp2_gpu, tp2_cpu, sizeof(int32_t) * width * height * 2, hipMemcpyHostToDevice);

hipMalloc((void **)&grid_dims_gpu, sizeof(int32_t) * 3);
hipMalloc((void **)&disparity_grid_gpu_1, sizeof(int32_t) * (param.disp_max + 2) * grid_height * grid_width);
hipMalloc((void **)&disparity_grid_gpu_2, sizeof(int32_t) * (param.disp_max + 2) * grid_height * grid_width);

hipMemcpy(grid_dims_gpu, grid_dims, sizeof(int32_t) * 3, hipMemcpyHostToDevice);
hipMemcpy(disparity_grid_gpu_1, disparity_grid_1, sizeof(int32_t) * (param.disp_max + 2) * grid_height * grid_width, hipMemcpyHostToDevice);
hipMemcpy(disparity_grid_gpu_2, disparity_grid_2, sizeof(int32_t) * (param.disp_max + 2) * grid_height * grid_width, hipMemcpyHostToDevice);



// init disparity image to -10
if (param.subsampling) {
    for (int32_t i = 0; i < (width / 2)*(height / 2); i++) {
        *(D1 + i) = -10;
        *(D2 + i) = -10;

    }
}
else {
    for (int32_t i = 0; i < width*height; i++) {
        *(D1 + i) = -10;
        *(D2 + i) = -10;
    }
}

//hipMalloc((void **)&p_support_gpu, sizeof(support_pt1) * P_SUPPORT_SIZE);
hipMalloc((void **)&tri_gpu_1, sizeof(triangle1) * TRI_SIZE1);
hipMalloc((void **)&tri_gpu_2, sizeof(triangle1) * TRI_SIZE2);
hipMalloc((void **)&D1_gpu, sizeof(float) * width * height);
hipMalloc((void **)&D2_gpu, sizeof(float) * width * height);
hipMalloc((void **)&P_gpu, sizeof(int32_t) * width * height);
hipMalloc((void **)&I_desc1, 16 * width*height * sizeof(uint8_t));
hipMalloc((void **)&I_desc2, 16 * width*height * sizeof(uint8_t));


hipMemcpy(tri_gpu_1, &tri_1[0], sizeof(Elas::triangle) * TRI_SIZE1, hipMemcpyHostToDevice);
hipMemcpy(tri_gpu_2, &tri_2[0], sizeof(Elas::triangle) * TRI_SIZE2, hipMemcpyHostToDevice);
hipMemcpy(D1_gpu, D1, sizeof(float) * width * height, hipMemcpyHostToDevice);
hipMemcpy(D2_gpu, D2, sizeof(float) * width * height, hipMemcpyHostToDevice);
hipMemcpy(I_desc1, I1, 16 * width*height * sizeof(uint8_t), hipMemcpyHostToDevice);
hipMemcpy(I_desc2, I2, 16 * width*height * sizeof(uint8_t), hipMemcpyHostToDevice);


// number of disparities
const int32_t disp_num = grid_dims[0] - 1;

// descriptor window_size
int32_t window_size = 2;

// pre-compute prior
float two_sigma_squared = 2 * param.sigma*param.sigma;
int32_t* P = new int32_t[disp_num];
for (int32_t delta_d = 0; delta_d<disp_num; delta_d++)
    P[delta_d] = (int32_t)((-log(param.gamma + exp(-delta_d*delta_d / two_sigma_squared)) + log(param.gamma)) / param.beta);
int32_t plane_radius = (int32_t)max((float)ceil(param.sigma*param.sradius), (float)2.0);
//plane_radius = 2;

hipMemcpy(P_gpu, P, sizeof(int32_t) * disp_num, hipMemcpyHostToDevice);

bool subsampling = param.subsampling;
int32_t match_texture = param.match_texture;
int32_t grid_size = param.grid_size;


err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(Triangle_Match1),hipFuncCachePreferL1);
if(hipSuccess != err)
{
    printf("hipFuncSetCacheConfig error %s\n", hipGetErrorString(err));
}
dim3 threads(320, 1);
dim3 grid(iDivUp(width, threads.x), iDivUp(height,threads.y));


//Triangle_Match1 << <grid, threads, 0, stream1>> > (tri_gpu_1, disparity_grid_gpu_1, \
//    grid_dims_gpu, I_desc1, I_desc2, P_gpu, plane_radius, 0, D1_gpu, width, height, TRI_SIZE1, subsampling, \
//                                                   match_texture, grid_size, tp1_gpu);

Triangle_Match1 << <grid, threads, 0, stream1>> > (tri_gpu_1, disparity_grid_gpu_1, \
    grid_dims_gpu, I_desc1, I_desc2, P_gpu, plane_radius, 0, D1_gpu, width, height, TRI_SIZE1, subsampling, \
                                                   match_texture, grid_size, tp1_gpu);


err = hipGetLastError();
if(hipSuccess != err)
{
    printf("error %s\n", hipGetErrorString(err));
}


err = hipFuncSetCacheConfig(reinterpret_cast<const void*>(Triangle_Match1),hipFuncCachePreferL1);
if(hipSuccess != err)
{
    printf("hipFuncSetCacheConfig error %s\n", hipGetErrorString(err));
}

//Triangle_Match1 << <grid, threads, 0, stream2>> > (tri_gpu_2, disparity_grid_gpu_2, \
//    grid_dims_gpu, I_desc1, I_desc2, P_gpu, plane_radius, 1, D2_gpu, width, height, TRI_SIZE2, subsampling, match_texture, grid_size,  tp2_gpu);
Triangle_Match1 << <grid, threads, 0, stream2>> > (tri_gpu_2, disparity_grid_gpu_2, \
    grid_dims_gpu, I_desc2, I_desc1, P_gpu, plane_radius, 1, D2_gpu, width, height, \
                     TRI_SIZE2, subsampling, match_texture, grid_size,  tp2_gpu);

 err = hipGetLastError();
if(hipSuccess != err)
{
    printf("error %s\n", hipGetErrorString(err));
}

hipMemcpy(D1, D1_gpu, sizeof(float) * width * height, hipMemcpyDeviceToHost);
hipMemcpy(D2, D2_gpu, sizeof(float) * width * height, hipMemcpyDeviceToHost);



}


//Triangle_Match1 << <1, 715*492>> > (tri_gpu_1, disparity_grid_gpu_1, \
//    grid_dims_gpu, I_desc1, I_desc2, P_gpu, plane_radius, 0, D1_gpu, width, height, TRI_SIZE1, subsampling, \
//                                                   match_texture, grid_size, tp1_gpu);




//for(int i = 0; i< 10000 ; i++)
//{
//    //cout <<I1+i<<" ";
//    printf("%d ", *(D1+i));
//    if(i%20 == 0)
//        cout<<endl;
//}


//    for(int i = 10000; i< 11000 ; i++)
//    {
//        //cout <<I1+i<<" ";
//        printf("%d ", *(D1+i));
//        if(i%20 == 0)
//            cout<<endl;
//    }





//printf("over memcpy\n");
// err = hipGetLastError();
//if(hipSuccess != err)
//{
//    printf("error %s\n", hipGetErrorString(err));
//}


//    for(int i = 7000; i< 8000 ; i++)
//    {
//        //cout <<I1+i<<" ";
//        printf("%d ", *(D1+i));
//        if(i%20 == 0)
//            cout<<endl;
//    }

//int main()
//{
//    cuda_computeD(int32_t* disparity_grid_1, int32_t* disparity_grid_2,  vector<Elas::support_pt> &p_support, \
//                  vector<Elas::triangle> &tri_1, vector<Elas::triangle> &tri_2, \
//                  float* D1, float* D2,uint8_t* I1, uint8_t* I2, int dim);
//    return 0;
//}
